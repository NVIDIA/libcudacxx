// Copyright (c) 2018-2020 NVIDIA Corporation
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// Released under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.

#include <cuda/std/cstdint>
#include <cuda/std/atomic>

// TODO: It would be great if this example could NOT depend on Thrust.
#include <thrust/pair.h>
#include <thrust/functional.h>
#include <thrust/allocate_unique.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/counting_iterator.h>

#include <cassert>
#include <random>

#include <iostream>
#include <cstdio>
#include <cassert>

// TODO: This should be upstreamed and then removed.
namespace thrust {

using universal_raw_memory_resource =
  thrust::system::cuda::detail::cuda_memory_resource<
    thrust::system::cuda::detail::hipMallocManaged, hipFree, void*
  >;

template <typename T>
using universal_allocator =
  thrust::mr::stateless_resource_allocator<T, universal_raw_memory_resource>;

template <typename T>
using universal_vector = thrust::device_vector<T, universal_allocator<T>>;

} // thrust

template <
  typename Key, typename Value,
  typename Hash     = thrust::identity<Key>,
  typename KeyEqual = thrust::equal_to<Key>,
  typename MemoryResource = thrust::universal_raw_memory_resource
>
struct concurrent_hash_table {
  // Elements transition from state_empty -> state_reserved ->
  // state_filled; no other transitions are allowed.
  enum state_type {
    state_empty, state_reserved, state_filled
  };

  using key_type       = Key;
  using mapped_type    = Value;
  using size_type      = cuda::std::uint64_t;

  using key_allocator    = thrust::mr::stateless_resource_allocator<
    key_type, MemoryResource
  >;
  using mapped_allocator = thrust::mr::stateless_resource_allocator<
    mapped_type, MemoryResource
  >;
  using state_allocator  = thrust::mr::stateless_resource_allocator<
    cuda::std::atomic<state_type>, MemoryResource
  >;

  using key_iterator   = typename key_allocator::pointer;
  using value_iterator = typename mapped_allocator::pointer;
  using state_iterator = typename state_allocator::pointer;

  // This whole thing is silly and should be a lambda, or at least a private
  // nested class, but alas, NVCC doesn't like that.
  struct element_destroyer {
  private:
    size_type      capacity_;
    key_iterator   keys_;
    value_iterator values_;
    state_iterator states_;

  public:
    __host__ __device__
    element_destroyer(size_type capacity,
                      key_iterator keys,
                      value_iterator values,
                      state_iterator states)
      : capacity_(capacity), keys_(keys), values_(values), states_(states)
    {}

    element_destroyer(element_destroyer const&) = default;

    __host__ __device__
    void operator()(size_type i) {
      if (state_empty != states_[i]) {
        (keys_ + i)->~key_type();
        (values_ + i)->~mapped_type();
      }
    }
  };

private:
  size_type      capacity_;
  key_iterator   keys_;
  value_iterator values_;
  state_iterator states_;
  Hash           hash_;
  KeyEqual       key_equal_;

public:
  __host__
  concurrent_hash_table(size_type capacity,
                        Hash hash = Hash(),
                        KeyEqual key_equal = KeyEqual())
    : capacity_(capacity)
    , keys_(key_allocator{}.allocate(capacity_))
    , values_(mapped_allocator{}.allocate(capacity_))
    , states_(state_allocator{}.allocate(capacity_))
    , hash_(std::move(hash))
    , key_equal_(std::move(key_equal))
  {
    thrust::uninitialized_fill(thrust::device,
                               states_, states_ + capacity_,
                               state_empty);
  }

  __host__
  ~concurrent_hash_table()
  {
    thrust::for_each(thrust::device,
                     thrust::counting_iterator<size_type>(0),
                     thrust::counting_iterator<size_type>(capacity_),
                     element_destroyer(capacity_, keys_, values_, states_));
  }

  // TODO: Change return type to an enum with three possible values, succeeded,
  // exists, and full.
  template <typename UKey, typename... Args>
  __host__ __device__
  thrust::pair<value_iterator, bool>
  try_emplace(UKey&& key, Args&&... args) {
    auto index{hash_(key) % capacity_};
    // Linearly probe the storage space up to `capacity_` times; if we haven't
    // succeeded by then, the container is full.
    for (size_type i = 0; i < capacity_; ++i) {
      state_type old = states_[index].load(cuda::std::memory_order_acquire);
      while (old == state_empty) {
        // As long as the state of this element is empty, attempt to set it to
        // reserved.
        if (states_[index].compare_exchange_weak(
              old, state_reserved, cuda::std::memory_order_acq_rel))
        {
          // We succeeded; the element is now "locked" as reserved.
          new (keys_ + index) key_type(std::forward<UKey>(key));
          new (values_ + index) mapped_type(std::forward<Args>(args)...);
          states_[index].store(state_filled, cuda::std::memory_order_release);
          return thrust::make_pair(values_ + index, true);
        }
      }
      // If we are here, the element we are probing is not empty and we didn't
      // fill it, so we need to wait for it to be filled.
      while (state_filled != states_[index].load(cuda::std::memory_order_acquire))
        ;
      // Now we know that the element we are probing has been filled by someone
      // else, so we check if our key is equal to it.
      if (key_equal_(keys_[index], key))
        // It is, so the element already exists.
        return thrust::make_pair(values_ + index, false);
      // Otherwise, the element isn't a match, so move on to the next element.
      index = (index + 1) % capacity_;
    }
    // If we are here, the container is full.
    return thrust::make_pair(value_iterator{}, false);
  }

  __host__ __device__
  mapped_type& operator[](key_type const& key) {
    return (*try_emplace(key).first);
  }
  __host__ __device__
  mapped_type& operator[](key_type&& key) {
    return (*try_emplace(std::move(key)).first);
  }
};

template <typename T>
struct identity_modulo {
private:
  T const modulo_;

public:
  __host__ __device__
  identity_modulo(T modulo) : modulo_(std::move(modulo)) {}

  identity_modulo(identity_modulo const&) = default;

  __host__ __device__
  T operator()(T i) { return i % modulo_; }
};

int main() {
  {
    using table = concurrent_hash_table<int, cuda::std::atomic<int>>;

    auto freq = thrust::allocate_unique<table>(thrust::universal_allocator<table>{}, 8);

    thrust::universal_vector<int> input = [] {
      thrust::universal_vector<int> v(2048);
      std::mt19937 gen(1337);
      std::uniform_int_distribution<long> dis(0, 7);
      thrust::generate(v.begin(), v.end(), [&] { return dis(gen); });
      return v;
    }();

    thrust::for_each(thrust::device, input.begin(), input.end(),
      [freq = freq.get()] __device__ (int i) {
        (*freq)[i].fetch_add(1, cuda::std::memory_order_relaxed);
      }
    );

    thrust::host_vector<int> gold(8);
    thrust::for_each(input.begin(), input.end(), [&] (int i) { ++gold[i]; });

    for (cuda::std::uint64_t i = 0; i < 8; ++i)
      std::cout << "i: " << i
                << " gold: " << gold[i]
                << " observed: " << (*freq)[i] << "\n";

    assert(hipSuccess == hipDeviceSynchronize());
  }
  {
    using table = concurrent_hash_table<int, cuda::std::atomic<int>, identity_modulo<int>>;

    auto freq = thrust::allocate_unique<table>(thrust::universal_allocator<table>{}, 8, identity_modulo<int>(4));

    thrust::universal_vector<int> input = [] {
      thrust::universal_vector<int> v(2048);
      std::mt19937 gen(1337);
      std::uniform_int_distribution<long> dis(0, 7);
      thrust::generate(v.begin(), v.end(), [&] { return dis(gen); });
      return v;
    }();

    thrust::for_each(thrust::device, input.begin(), input.end(),
      [freq = freq.get()] __device__ (int i) {
        (*freq)[i].fetch_add(1, cuda::std::memory_order_relaxed);
      }
    );

    thrust::host_vector<int> gold(8);
    thrust::for_each(input.begin(), input.end(), [&] (int i) { ++gold[i]; });

    for (cuda::std::uint64_t i = 0; i < 8; ++i)
      std::cout << "i: " << i
                << " gold: " << gold[i]
                << " observed: " << (*freq)[i] << "\n";

    assert(hipSuccess == hipDeviceSynchronize());
  }
}

